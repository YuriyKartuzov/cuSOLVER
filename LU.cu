#include "LU.h"
using namespace std;
using namespace chrono;

// gpuLU
double gpuLU(int N, bool debug) {
	steady_clock::time_point timeStart, timeEnd;// variables for timing
	hipblasStatus_t stat;
	hipError_t cudaStatus;
	hipsolverStatus_t cusolverStatus;
	hipsolverHandle_t handle;

	// Host Variables
	double *A, *B1, *B; 

	// Device variables
	double *d_A, *d_B, *d_Work;
	int * d_pivot, *d_info, Lwork;
	int info_gpu = 0;
	
	// PART 1 ---------------------------------------------------------------------------------------------
	// allocate memory, generate random numbers for the matrix
	A = (double *)malloc(N*N * sizeof(double));
	B = (double *)malloc(N * sizeof(double));
	B1 = (double *)malloc(N * sizeof(double));
	for (int i = 0; i<N*N; i++) A[i] = rand() / (double)RAND_MAX;   // Randomize A
	for (int i = 0; i<N; i++) B[i] = 0.0;                           // initialize B
	for (int i = 0; i<N; i++) B1[i] = 1.0;                          // Initialize B1
	double al = 1.0, bet = 0.0;                                     // setting coefficientimeStart cblas_dgemv function
	int incx = 1, incy = 1;

	// TIMER start
	timeStart = steady_clock::now();

	cblas_dgemv(CblasColMajor, CblasNoTrans, N, N, al, A, N, B1, incx, bet, B, incy); // multiply B=A*B1

	// TIMER end															 
	auto cblasRUN = steady_clock::now() - timeStart;
	
	// PART 2 ---------------------------------------------------------------------------------------------
	// prepare memory on the device
	cudaStatus = hipGetDevice(0);
	cusolverStatus = hipsolverDnCreate(&handle);
	cudaStatus = hipMalloc((void **)& d_A, N * N * sizeof(double));
	cudaStatus = hipMalloc((void **)& d_B, N * sizeof(double));
	cudaStatus = hipMalloc((void **)& d_pivot, N * sizeof(int));
	cudaStatus = hipMalloc((void **)& d_info, sizeof(int));

	cudaStatus = hipMemcpy(d_A, A, N*N * sizeof(double), hipMemcpyHostToDevice); // copy d_A <-A
	cudaStatus = hipMemcpy(d_B, B, N * sizeof(double), hipMemcpyHostToDevice);   // copy d_B <-B

	// compute buffer size and prep memory
	cusolverStatus = hipsolverDnDgetrf_bufferSize(handle, N, N, d_A, N, &Lwork);
	cudaStatus = hipMalloc((void **)& d_Work, Lwork * sizeof(double));

	// START timer
	timeStart = steady_clock::now();

	// MAIN FUNCTIONS
	cusolverStatus = hipsolverDnDgetrf(handle, N, N, d_A, N, d_Work, d_pivot, d_info);
	cusolverStatus = hipsolverDnDgetrs(handle, HIPBLAS_OP_N, N, 1, d_A, N, d_pivot, d_B, N, d_info);
	cudaStatus = hipDeviceSynchronize();

	// END timer
	auto cusolverRUN = steady_clock::now() - timeStart;

	// Main output
	if (debug) {
		cudaStatus = hipMemcpy(&info_gpu, d_info, sizeof(int), hipMemcpyDeviceToHost); // d_info -> info_gpu
		printf("\n  Run successfull: ", info_gpu);
		cudaStatus = hipMemcpy(B1, d_B, N * sizeof(double), hipMemcpyDeviceToHost);    // d_B ->B1
		printf(" Solution : ");
		for (int i = 0; i < 5; i++) printf("%g, ", B1[i]);
		printf(" ... "); // print first componentimeStart of the solution
		printf("\n");
	}

	// Garbage Collection
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_pivot);
	hipFree(d_info);
	hipFree(d_Work);
	free(A); free(B); free(B1);
	cusolverStatus = hipsolverDnDestroy(handle);
	//hipDeviceReset();

	//system("pause"); // needed to executable
	double returnVal = duration_cast<milliseconds>(cusolverRUN).count();
	return returnVal;
}






